#include "hip/hip_runtime.h"
#include"total_include.cuh"
#include"scma_cuda_test.cuh" 
#include"uniform.cuh"
#include"gaussian.cuh"
#include"space_allocate.cuh"
#include"scma_cuda_utils.cuh"
//#include"scma_decode_serial.cuh"

#include"scma_decode_cuda_v3.cuh"

void main(){
	hipDeviceReset();
	//MemFree_test();
	//bias_test();

	hipStream_t    stream0, stream1;

	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	size_t total_mem, free_mem;

	hipComplex*** CB = get_default_codebook();

	const int SNR_len = 6;
	double SNR_dB[SNR_len] = { 8, 10, 12, 14, 16, 18 };

	int Nerr1[SNR_len] = { 0, 0, 0, 0, 0, 0 };
	int Nerr2[SNR_len] = { 0, 0, 0, 0, 0, 0 };

	double BER_avg1[SNR_len] = { 0, 0, 0, 0, 0, 0 };
	double BER_avg2[SNR_len] = { 0, 0, 0, 0, 0, 0 };

	int N_b = 384;   //length of  bit sequence 

	int N_s = N_b / log2(M); //length of symbol sequence

	int f_num = 5;

	int** Bits1 = allocate_Bits_host(N_b);
	int** Symbols1 = allocate_Bits_host(N_s);
	int** b_dec1_dev = allocate_Bits_dev(N_b);
	int** s_dec1_dev = allocate_Symbols_dev(N_s);


	int** Bits2 = allocate_Bits_host(N_b);
	int** Symbols2 = allocate_Bits_host(N_s);
	int** b_dec2_dev = allocate_Bits_dev(N_b);
	int** s_dec2_dev = allocate_Symbols_dev(N_s);

	hipComplex*** h = allocate_h(N_s);             //size(h) =[J][N][K]

	hipComplex*** x1 = allocate_x_host(N_s);           //size(x) =[J][N][K]
	hipComplex** y1 = allocate_y(N_s);

	hipComplex*** x2 = allocate_x_host(N_s);         
	hipComplex** y2 = allocate_y(N_s);

	float**** Igv1 = allocatef_Igv(N_s);
	float**** Ivg1 = allocatef_Ivg(N_s);

	float**** Igv2 = allocatef_Igv(N_s);
	float**** Ivg2 = allocatef_Ivg(N_s);

	int** Ind_J = get_J_indices(CB, Dc);
	int** Ind_K = get_K_indices(CB, Dr);

	float   total_elapsedTime = 0;

	for (int s = 0; s < 6; s++){
		double p_noise = 1 / pow(10, SNR_dB[s] / 10);
		for (int f = 0; f < f_num; f++){
			//Igv_Ivg_init(Igv, Ivg, N_s);

			hipEvent_t     start, stop;
			float           elapsedTime;

			hipEventCreate(&start);
			hipEventCreate(&stop);

			//hipEventRecord(start, 0);

			source_bit(Bits1, N_b);
			source_bit(Bits2, N_b);

			get_h(h, N_s);
			scma_encode(x1, CB, Bits1, Symbols1, N_s);
			scma_uplink_transmisson(y1, h, x1, N_s, p_noise);
			
			scma_encode(x2, CB, Bits2, Symbols2, N_s);
			scma_uplink_transmisson(y2, h, x2, N_s, p_noise);

			hipEventRecord(start, 0);

			scma_decode_main_kernel << <1, 1, 0, stream0 >> >(b_dec1_dev, s_dec1_dev, y1, h, CB, N_s, p_noise, Igv1, Ivg1, Ind_K, Ind_J);
			scma_decode_main_kernel << <1, 1, 0, stream1 >> >(b_dec2_dev, s_dec2_dev, y2, h, CB, N_s, p_noise, Igv2, Ivg2, Ind_K, Ind_J);

			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsedTime, start, stop);
			printf("Time taken:  %3.1f ms\n", elapsedTime);
			total_elapsedTime += elapsedTime;

			hipDeviceSynchronize();
			hipStreamSynchronize(stream0);
			hipStreamSynchronize(stream1);

			for (int j = 0; j < J; j++){
				for (int nb = 0; nb < N_b; nb++){
					if (b_dec1_dev[j][nb] != Bits1[j][nb]){
						Nerr1[s]++;
					}
					if (b_dec2_dev[j][nb] != Bits2[j][nb]){
						Nerr2[s]++;
					}
				}
			}

			//printf("%d\n",Nerr[s]);

			//free_variables22(x, h, y, N_s);

			//hipMemGetInfo(&free_mem, &total_mem);
			//std::cout << " Currently " << free_mem << " bytes free" << std::endl;

		}
		BER_avg1[s] = (double)Nerr1[s] / (f_num * N_b * J);
		BER_avg2[s] = (double)Nerr2[s] / (f_num * N_b * J);
		cout << "average BER1 of SNR =" << SNR_dB[s] << "dB: " << BER_avg1[s] << endl;
		cout << "average BER2 of SNR =" << SNR_dB[s] << "dB: " << BER_avg2[s] << endl;
	}

	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);

	printf("total decoding throughtput:%f Kb/s", (2 * SNR_len*f_num*N_b*J) / total_elapsedTime);
	system("pause");

}